
#include <hip/hip_runtime.h>
#include <cudnn.h>

__global__ void powf_global(float x, float y, float *r)
{
    *r = __powf(x, y);
}

float cuda_pow(float x, float y)
{

    float *gpu_result, result;
    hipMalloc((void **)&gpu_result, sizeof(float));
    powf_global<<<1, 1>>>(x, y, gpu_result);
    hipMemcpy(&result, gpu_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpu_result);
    // cudaDeviceReset(); // force printf flush
    return result;
}
