
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudnn.h"

__global__ void cuda_pow_g(float x, float y) {
    auto r = __powf(x, y);
    printf("pow(%f, %f) == %f\n", x, y, r);
}

void cuda_pow(float x, float y)
{
    cuda_pow_g<<<1, 1>>>(x, y);
    hipDeviceReset();
}
