
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudnn.h"

__global__ void cuda_pow_gpu(float x, float y) {
    auto r = __powf(x, y);
    printf("pow(%f, %f) == %f\n", x, y, r);
}

void cuda_pow(float x, float y)
{
    cuda_pow_gpu<<<1, 1>>>(x, y);
    hipDeviceReset();
}
