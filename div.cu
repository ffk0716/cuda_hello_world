
#include <hip/hip_runtime.h>
#include <cudnn.h>

__global__ void fdiv_rn_global(float x, float y, float *r)
{
    *r = __fdiv_rn(x, y);
}

float cuda_fdiv_rn(float x, float y)
{

    float *gpu_result, result;
    hipMalloc((void **)&gpu_result, sizeof(float));
    fdiv_rn_global<<<1, 1>>>(x, y, gpu_result);
    hipMemcpy(&result, gpu_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpu_result);
    // cudaDeviceReset(); // force printf flush
    return result;
}

__global__ void fdividef_global(float x, float y, float *r)
{
    *r = __fdividef(x, y);
}

float cuda_fdividef(float x, float y)
{

    float *gpu_result, result;
    hipMalloc((void **)&gpu_result, sizeof(float));
    fdividef_global<<<1, 1>>>(x, y, gpu_result);
    hipMemcpy(&result, gpu_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpu_result);
    // cudaDeviceReset(); // force printf flush
    return result;
}
